#include "hip/hip_runtime.h"
#include "MatrixNaiveGPUCuda.h"

#define NOMINMAX
#include <algorithm>

unsigned long closest_power2(unsigned long v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

#ifdef __HIPCC__
__global__ void matrix_mult_kernel(double *u, double *v, double *ret,
                                   int n, int p) {

    const int r = blockIdx.x * blockDim.x + threadIdx.x;
    const int c = blockIdx.y * blockDim.y + threadIdx.y;

    //printf("sr=%d sc=%d (bx=%d, by=%d)\n", r, c, blockDim.x, blockDim.y);

    if (r < n && c < p)
        for(int i = 0; i < n; ++i) {
            ret[c + r * p] += u[i + r * n] * v[c + i * p];
            //printf("(%d %d) u[%d](%4.2f) * v[%d](%4.2f) +=> %4.2f\n", r, c, i + r * n, u[i + r * n],  c+ i * p, v[c + i * p], ret[c + r * p]);
        }
}

void check_cuda_error(hipError_t e){
    if(e != hipSuccess) {
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__, hipGetErrorString(e));
    }
}

void check_cuda_error(){
    check_cuda_error(hipGetLastError());
}


int get_block_num(int n, int bsize){
    return (n + bsize - 1) / bsize;
}

MatrixNaiveCUDA MatrixNaiveCUDA::operator*(MatrixNaiveCUDA &v) {
    assert(u.cols() == v.rows());
    MatrixNaiveCUDA r(rows(), v.cols(), 0);

    /*
    DEBUG("Calling Cuda Code\n");
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    DEBUG("MaxGridSize      : %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    DEBUG("MaxThreadsDim    : %d x %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    DEBUG("MaxThreadPerBlock: %d\n", prop.maxThreadsPerBlock);*/

    // unsigned long closest_power2(unsigned long v)
    size_t xblock_size  = std::min(unsigned long long (32), rows());
    size_t yblock_size  = std::min(unsigned long long (32), cols());
    dim3 threadsPerBlock(xblock_size, yblock_size);
    dim3 numBlocks(get_block_num(r.rows(), xblock_size), get_block_num(r.cols(), yblock_size));

    DEBUG("threadsPerBlock (%lu x %lu)\n", threadsPerBlock.x, threadsPerBlock.y);
    DEBUG("numBlocks (%lu x %lu)\n", numBlocks.x, numBlocks.y);

    hipError_t e = hipGetLastError();

    check_cuda_error();
    matrix_mult_kernel<<<numBlocks, threadsPerBlock>>>(&elems[0], &v.elems[0], &r.elems[0], v.rows(),
                                               v.cols());
    check_cuda_error();
    hipDeviceSynchronize();
    return r;
}
MatrixNaiveCUDA::MatrixNaiveCUDA(size_t r, size_t c, double val) : vrows(r), _rows(r), _cols(c) {

    // elems(r * c, val),
    DEBUG("TEST\n");
    _true_rows = closest_power2(rows());
    _true_cols = closest_power2(cols());
    size_t bytes = std::max(unsigned long long (256), _true_rows * _true_cols * sizeof(double));

    DEBUG("%llu %llu\n", _true_rows, _true_cols);
    DEBUG("Allocate %4.2f Mo %llu\n", float(bytes) / (1024.0 * 1024.0), bytes / sizeof(double));

    check_cuda_error(hipMallocManaged(&elems, bytes));

    for(size_t i = 0; i < size(); ++i) {
        elems[i] = val;
    }
    for(size_t i = size(); i < bytes / sizeof(double); ++i) {
        elems[i] = 0;
    }

    DEBUG("Initialization\n");
    for(size_t i = 0; i < rows(); ++i) {
        vrows[i] = &elems[i * cols()];
    }
}

MatrixNaiveCUDA::~MatrixNaiveCUDA() { hipFree(elems); }

MatrixNaiveCUDA::MatrixNaiveCUDA(MatrixNaiveCUDA const &data) :
    vrows(data.vrows), _rows(data._rows), _cols(data._cols) {
    DEBUG("Copying");
    size_t bytes = std::max(unsigned long (256), closest_power2(size() * sizeof(double)));
    check_cuda_error(hipMallocManaged(&elems, bytes));

    for(size_t i = 0; i < size(); ++i) {
        elems[i] = data.elems[i];
    }
    for(size_t i = size(); i < bytes / sizeof(double); ++i) {
        elems[i] = 0;
    }

    for(size_t i = 0; i < rows(); ++i) {
        vrows[i] = &elems[i * cols()];
    }
}
#endif
